#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime_api.h>
#include <time.h>

__device__ int is_a_match(char *attempt) {
  char password1[] = "AV70";
  char password2[] = "ES07";
  char password3[] = "HT12";
  char password4[] = "RB23";

  char *s = attempt;
  char *u = attempt;
  char *j = attempt;
  char *n = attempt;
  char *pwd1 = password1;
  char *pwd2 = password2;
  char *pwd3 = password3;
  char *pwd4 = password4;

  while(*s == *pwd1) {
   if(*s == '\0')
    {
    printf("Found password: %s\n",password1);
      break;
    }

    s++;
    pwd1++;
  }
    
  while(*u == *pwd2) {
   if(*u == '\0')
    {
    printf("Found password: %s\n",password2);
      break;
}

    u++;
    pwd2++;
  }

  while(*j == *pwd3) {
   if(*j == '\0')
    {
    printf("Found password: %s\n",password3);
      break;
    }

    j++;
    pwd3++;
  }

  while(*n == *pwd4) {
   if(*n == '\0')
    {
    printf("Found password: %s\n",password4);
      return 1;
    }

    n++;
    pwd4++;
  }
  return 0;

}


__global__ void  kernel() {
char t,k;
 
  char password[5];
  password[4] = '\0';

int i = blockIdx.x+65;
int j = threadIdx.x+65;
char firstValue = i;
char secondValue = j;
    
password[0] = firstValue;
password[1] = secondValue;
    for(t='0'; t<='9'; t++){
      for(k='0'; k<='9'; k++){
            password[2] = t;
            password[3] = k;
          if(is_a_match(password)) {
        //printf("Success");
          }
             else {
         //printf("tried: %s\n", password);          
            }
          }
        } 
      
}
int time_difference(struct timespec *start,
                    struct timespec *finish,
                    long long int *difference) {
  long long int ds =  finish->tv_sec - start->tv_sec;
  long long int dn =  finish->tv_nsec - start->tv_nsec;

  if(dn < 0 ) {
    ds--;
    dn += 1000000000;
  }
  *difference = ds * 1000000000 + dn;
  return !(*difference > 0);
}


int main() {

  struct  timespec start, finish;
  long long int time_elapsed;
  clock_gettime(CLOCK_MONOTONIC, &start);

kernel <<<26,26>>>();
  hipDeviceSynchronize();

  clock_gettime(CLOCK_MONOTONIC, &finish);
  time_difference(&start, &finish, &time_elapsed);
  printf("Time elapsed was %lldns or %0.9lfs\n", time_elapsed, (time_elapsed/1.0e9));
  return 0;
}




